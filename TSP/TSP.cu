
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <limits.h>

int rand_range(int min, int max);
int* make_pts(int N);

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s N\n", argv[0]);
        exit(-1);
    }

    long int N = strtol(argv[1], NULL, 10);
    if (N <= 0) {
        printf("Please enter a positive int for N\n");
        exit(-1);
    }
    else if (N == LONG_MAX || N == LONG_MIN) {
        printf("The provided N is too %s.\n", N == LONG_MAX ? "large" : "small");
        exit(-1);
    }

    printf("N: %d\n", N);

    int* pts = make_pts(N);

    for (int i = 0; i < 2 * N; i+=2)
        printf("%d: (%d, %d)\n", i / 2, pts[i], pts[i + 1]);

    int* perms = (int*) malloc(sizeof(int) * N);
    for (int i = 0; i < N; i++) perms[i] = i;

    free(pts);
    free(perms);
    return 0;
}

int rand_range(int min, int max) {
    double u = rand() / (double)RAND_MAX;
    return (max - min + 1) * u + min;
}

int* make_pts(int N) {
    int* pts = (int*) malloc(sizeof(int) * N * 2);
    srand(time(NULL));
    for (int i = 0; i < 2 * N; i+=2) {
        pts[i] = rand_range(0, 1000);
        pts[i + 1] = rand_range(0, 1000);
    }

    return pts;
}
