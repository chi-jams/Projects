
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <math.h>

#include <algorithm>

int rand_range(int min, int max);
int* make_pts(int N);
int* gen_perm(int n, int k);

double get_dist(const int* pts, int i, int j);

void serial_tsp(int* pts, double& min_dist, int& min_perm, int N, int Nf);

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s N\n", argv[0]);
        exit(-1);
    }

    long int N = strtol(argv[1], NULL, 10);
    if (N <= 0) {
        printf("Please enter a positive int for N\n");
        exit(-1);
    }
    else if (N == LONG_MAX || N == LONG_MIN) {
        printf("The provided N is too %s.\n", N == LONG_MAX ? "large" : "small");
        exit(-1);
    }

    printf("N: %d\n", N);
    unsigned long long int Nf = 1;
    for (int i=N;i>0;i--) Nf*=i;

    int* pts = make_pts(N);

    for (int i = 0; i < 2 * N; i+=2)
        printf("%d: (%d, %d)\n", i / 2, pts[i], pts[i + 1]);

    double min_dist;
    int min_perm;
    serial_tsp(pts, min_dist, min_perm, N, Nf);
    printf("Min distance: %f\n", min_dist);
    printf("Path:\n");
    int* perm = gen_perm(N, min_perm);
    for (int i = 0; i < N; i++) {
        int p_i = perm[i];
        printf("%d: (%d, %d)\n", p_i, pts[2*p_i], pts[2*p_i + 1]);
    }

    free(pts);
    return 0;
}

int rand_range(int min, int max) {
    double u = rand() / (double)RAND_MAX;
    return (max - min + 1) * u + min;
}

int* make_pts(int N) {
    int* pts = (int*) malloc(sizeof(int) * N * 2);
    srand(time(NULL));
    for (int i = 0; i < 2 * N; i+=2) {
        pts[i] = rand_range(0, 1000);
        pts[i + 1] = rand_range(0, 1000);
    }

    return pts;
}

int* gen_perm(int n, int perm) {
    int i, ind, m=perm;
    int* p = (int*) malloc(sizeof(int) * n);
    int* e = (int*) malloc(sizeof(int) * n);

    for (i=0;i<n;i++)e[i]=i;
    for (i=0;i<n;i++) {
        ind = m % (n - i);
        m = m / (n - i);
        p[i] = e[ind];
        e[ind] = e[n - i - 1];
    }

    free(e);

    return p;
}

double get_dist(const int* pts, int i, int j) {
    int x1 = pts[2*i],
        y1 = pts[2*i + 1],
        x2 = pts[2*j],
        y2 = pts[2*j + 1];

    int dx = x1 - x2,
        dy = y1 - y2;
    dx *= dx;
    dy *= dy;
    
    return sqrt(dx + dy);
}

void serial_tsp(int* pts, double& min_dist, int& min_perm, int N, int Nf) {
    min_dist = -1;
    int* perm;

    for (int i = 0; i < Nf; i++) {
        perm = gen_perm(N, i); 

        double dist = 0;
        for (int j = 0; j < N; j++) {
            dist += get_dist(pts, perm[j], perm[(j+1) % N]);
        }

        if (dist < min_dist || min_dist < 0) {
            min_dist = dist;
            min_perm = i;
        }
        
        free(perm);
    }
}
